
#include <hip/hip_runtime.h>
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main()
{
    long int N = 10000;
    size_t size = N * sizeof(float);
    long int i = 0;

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    for (i = 0; i < N; i++) {
         h_A[i] = i*0.5;
         h_B[i] = i*0.25;
    }

    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);
    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    VecAdd<<<1, 100>>>(d_A, d_B, d_C, N);

     // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
